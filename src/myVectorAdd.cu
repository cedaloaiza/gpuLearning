//Defines the CUDA API functions and built-in variables
#include<hip/hip_runtime.h>

#include <iostream>


void myVecAddSec(float* A, float* B, float* C, int n){
	for(int i = 0; i < n; i++){
		C[i] = A[i] + B[i];
	}

}

// Compute vector sum C = A+B
// Each thread performs one pair-wise addition
//Executed on the device, and only callable from the host
__global__
void vecAddKernel(float* A, float* B, float* C, int n){
	//threadIdx.x, blockDim.x, blockIdx.x are hardware registers. could be ignore this line?
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	if(i<n) C[i] = A[i] + B[i];
}

void vecAdd(float* A, float*B, float* C, int n){
	int size = n* sizeof(float);
	float *d_A, *d_B, *d_C;

	/*
	Part 1:
	 Allocate device memory for A, B, and C
	 copy A and B to device 	memory

	– Allocates object in the device global memory
	– Two parameters
	• Address of a pointer to the allocated object
	• Size of allocated object in terms of bytes
	*/
	hipMalloc((void ** )&d_A, size);
	hipMalloc((void **)&d_B,size);
	hipMalloc((void **)&d_C,size);
	/*
	cudaMemcpy()
	– memory data transfer
	– Requires four parameters
	• Pointer to destination
	• Pointer to source
	• Number of bytes copied
	• Type/Direction of transfer
	*/
	hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);



	/*
	Part 2:
	 Kernel launch code – to have the device
	 to perform the actual vector addition
	The first configuration parameter gives the number of thread blocks in the grid.
	The second specifies the number of threads in each thread block.
	*/
	vecAddKernel<<<ceil(n/256.0), 256>>> (d_A, d_B, d_C, n);

	//Part 3:
	// copy C from the device memory (Global Memory)
	// Free device vectors
	hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
	//free the storage space for the vector from the device global memory
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
}



int main(){
	int n = 700000;
	float A[700000];// = {1,2};
	float B[700000];// = {4,5};
	float C[n];

	for(int i = 0; i < n; ++i)
	{
	  A[i] = rand() % 80 + 1;
	  B[i] = rand() % 80 + 1;
	}
	long startTime = (unsigned long)time(NULL);

	//Sequential
	//myVecAddSec(A,B,C,n);
	//CUDA
	vecAdd(A,B,C,n);
	long finishTime = (unsigned long)time(NULL);

	std::cout<<"Sum between "<<A[51]<<" and "<<B[51]<<" : "<<C[51]<<"\n";
	std::cout<<"computing time: "<<finishTime-startTime;
	return 0;
}
