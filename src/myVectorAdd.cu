#include<hip/hip_runtime.h>
#include <iostream>

void myVecAdd(float* A, float* B, float* C, int n){
	for(int i = 0; i < n; i++){
		C[i] = A[i] + B[i];
	}

}

int main(){
	int n = 2;
	float A[2] = {1,2};
	float B[2] = {4,5};
	float C[n];
	myVecAdd(A,B,C,n);
	std::cout<<"hola "<<C[0];

	return 0;
}
